#include "hip/hip_runtime.h"
// 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

// Simple transformation kernel
__global__ void transformKernel(float* output,
                                int width, int height,
                                float theta) 
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    // Transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;


    // Read from texture and write to global memory
    output[y * width + x] = tex2D(texRef, tu, tv);
}

// Host code
int main()
{
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // Copy to device memory some data located at address h\_data
    // in host memory 
    hipMemcpyToArray(cuArray, 0, 0, h_data, size,
                      hipMemcpyHostToDevice);

    // Set texture reference parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode     = hipFilterModeLinear;
    texRef.normalized     = true;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRef, cuArray, channelDesc);

    // Allocate result of transformation in device memory
    float* output;
    hipMalloc(&output, width * height * sizeof(float));

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    transformKernel<<<dimGrid, dimBlock>>>(output, width, height,
                                           angle);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);

    return 0;
}

