#include "hip/hip_runtime.h"
// 2D surfaces
surface<void, 2> inputSurfRef;
surface<void, 2> outputSurfRef;
            
// Simple copy kernel
__global__ void copyKernel(int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfRef, x * 4, y);
        // Write to output surface
        surf2Dwrite(data, outputSurfRef, x * 4, y);
    }
}

// Host code
int main()
{
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
             hipCreateChannelDesc(8, 8, 8, 8,
                                   hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Copy to device memory some data located at address h\_data
    // in host memory 
    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size,
                      hipMemcpyHostToDevice);

    // Bind the arrays to the surface references
    cudaBindSurfaceToArray(inputSurfRef, cuInputArray);
    cudaBindSurfaceToArray(outputSurfRef, cuOutputArray);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    copyKernel<<<dimGrid, dimBlock>>>(width, height);


    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);

    return 0;
}

